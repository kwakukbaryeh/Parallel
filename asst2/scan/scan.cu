#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"


extern float toBW(int bytes, float sec);


/* Helper function to round up to a power of 2.
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void upsweep_kernel(int* data, int N, int twod) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = twod * 2;

    if (index < N / stride) {
        int offset = index * stride + twod - 1;
        data[offset + twod] += data[offset];
    }
}

__global__ void downsweep_kernel(int* data, int N, int twod) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = twod * 2;

    if (index < N / stride) {
        int offset = index * stride + twod - 1;
        int t = data[offset];
        data[offset] = data[offset + twod];
        data[offset + twod] += t;
    }
}

void exclusive_scan(int* device_data, int length)
{
    /* TODO
     * Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the data in device memory
     * The data are initialized to the inputs.  Your code should
     * do an in-place scan, generating the results in the same array.
     * This is host code -- you will need to declare one or more CUDA
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the data array is sized to accommodate the next
     * power of 2 larger than the input.
     */
    int N = nextPow2(length);

    // Upsweep Phase
    for (int twod = 1; twod < N; twod *= 2) {
        int blocks = (N / (2 * twod) + 255) / 256;
        upsweep_kernel<<<blocks, 256>>>(device_data, N, twod);
        hipDeviceSynchronize();
    }

    // Set root to 0 for exclusive scan
    hipMemset(&device_data[N - 1], 0, sizeof(int));

    // Downsweep Phase
    for (int twod = N / 2; twod >= 1; twod /= 2) {
        int blocks = (N / (2 * twod) + 255) / 256;
        downsweep_kernel<<<blocks, 256>>>(device_data, N, twod);
        hipDeviceSynchronize();
    }
}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness.
    // You may have an easier time in your implementation if you assume the
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    hipMalloc((void **)&device_data, sizeof(int) * rounded_length);

    hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;

    hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}

__global__ void mark_peaks_kernel(int* input, int* markers, int length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Correct boundary check
    if (index > 0 && index < length - 1) {
        if (input[index] > input[index - 1] && input[index] > input[index + 1]) {
            markers[index] = 1;
        } else {
            markers[index] = 0;
        }
    }

    // Clear out-of-bounds elements if padded to next power of 2
    if (index >= length) {
        markers[index] = 0;
    }
}

__global__ void gather_peaks_kernel(int* markers, int* scan_result, int* output, int length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < length && markers[index] == 1) {
        int output_index = scan_result[index];
        if (output_index < length) {
            output[output_index] = index;  // Safeguard against overflows
        }
}
    
    if (index >= length) {
        return;
    }
}

int find_peaks(int *device_input, int length, int *device_output) {
    /* TODO:
     * Finds all elements in the list that are greater than the elements before and after,
     * storing the index of the element into device_result.
     * Returns the number of peak elements found.
     * By definition, neither element 0 nor element length-1 is a peak.
     *
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if
     * it requires that. However, you must ensure that the results of
     * find_peaks are correct given the original length.
     */
    int rounded_length = nextPow2(length);
    int *markers, *scan_result;

    // Allocate memory with rounded size
    hipMalloc(&markers, sizeof(int) * rounded_length);
    hipMalloc(&scan_result, sizeof(int) * rounded_length);

    // Mark peaks
    int blocks = (length + 255) / 256;
    mark_peaks_kernel<<<blocks, 256>>>(device_input, markers, length);
    hipDeviceSynchronize();

    // Clear padded elements in markers to 0
    int zero_size = rounded_length - length;
    if (zero_size > 0) {
        hipMemset(markers + length, 0, zero_size * sizeof(int));
    }

    // Perform exclusive scan on the rounded markers array
    hipMemcpy(scan_result, markers, sizeof(int) * rounded_length, hipMemcpyDeviceToDevice);
    exclusive_scan(scan_result, rounded_length);

    // Gather peaks using original length
    gather_peaks_kernel<<<blocks, 256>>>(markers, scan_result, device_output, length);
    hipDeviceSynchronize();

    // Get the total number of peaks from the end of the scan_result
    int num_peaks;
    hipMemcpy(&num_peaks, &scan_result[rounded_length - 1], sizeof(int), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(markers);
    hipFree(scan_result);

    return num_peaks;
}



/* Timing wrapper around find_peaks. You should not modify this function.
 */
double cudaFindPeaks(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    int result = find_peaks(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return endTime - startTime;
}


void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
